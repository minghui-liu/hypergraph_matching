#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <limits.h>

#pragma once
#define BLOCK_SIZE 32
#define BLOCK_SIZE_DIM1 1024

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
	double* elements;
} Matrix;

//function to print a matrix
void printMatrix(Matrix A) {
	printf("\n");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			printf("%.4f ", A.elements[i*A.width+j]); 
		}
		printf("\n");
	}
	printf("\n");
}

// function to save matrix in an ascii file */
void saveMatrix(Matrix A, char *filename) {
	FILE *fp;
	fp = fopen(filename, "w");
	for (int i=0; i<A.height; i++) {
		for (int j=0; j<A.width; j++) {
			fprintf(fp, "%.4f ", A.elements[i*A.width+j]); 
		}
		fprintf(fp, "\n");
	}
	fclose(fp);
}


/*** matlib functions listed in alphabetical order *****/


/************************ G ************************/
// matrix getCol kernel
__global__
void getColKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(col == num) 
		d_Out.elements[row] = d_In.elements[row*d_In.width+col];
}

void getCol(Matrix In, Matrix Out, int num) {
	//printf("getCol()\n");
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	//printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  err = hipMalloc(&d_Out.elements, size);
  //printf("CUDA malloc Out: %s\n", hipGetErrorString(err));	

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	getColKernel<<<dimGrid, dimBlock>>>(d_In, d_Out, num);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// matrix getRow kernel
__global__
void getRowKernel(Matrix d_In, Matrix d_Out, int num) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	if(row == num) 
		d_Out.elements[col] = d_In.elements[row*d_In.width+col];
}

/************************ I ************************/

// check if a square matrix is symmetric
__global__
void isSymmetricKernel(Matrix d_A, int *d_result) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	if(d_A.elements[row*d_A.width+col] != d_A.elements[row + col*d_A.width])
		*(d_result) = 0;
}

int isSymmetric(Matrix A) {
	//printf("isSymmetric()\n");
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	//printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	//printf("Copy A to device: %s\n", hipGetErrorString(err));

	// load result to device memory
	int result = 1;
	int *d_result;
	err = hipMalloc(&d_result, sizeof(int));
	//printf("CUDA malloc d_result: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);	
	//printf("Copy result to device: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	isSymmetricKernel<<<dimGrid, dimBlock>>>(d_A, d_result);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	//read result from fdevice memory
	err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
	//printf("Copy result off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_result);

	return result;
}

/************************ M ************************/

// matAdd kernel
__global__
void matAddKernel(Matrix d_A, Matrix d_B, Matrix d_C) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_C.elements[row*d_C.width + col] = d_A.elements[row*d_A.width + col] + d_B.elements[row*d_B.width + col];
}

// matrix matDiv kernel called by matDiv()
__global__
void matDivKernel(Matrix d_A, Matrix d_B, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row*d_A.width+col;
	if(row > d_A.height || col > d_A.width) return;
	d_Out.elements[idx] = d_A.elements[idx] / d_B.elements[idx];
}

void matDiv(Matrix A, Matrix B, Matrix Out) {
	//printf("matDiv()\n");
	if (A.width != B.width || A.height != B.height) {
		printf("Input matrices must have the same dimension!\n");
		return;
	}
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	//printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix A to device: %s\n", hipGetErrorString(err));
	
	// load B to device memory
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	err = hipMalloc(&d_B.elements, size);
	//printf("CUDA malloc B: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix B to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matDivKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_Out);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_Out.elements);

}

// matrix matPlusScaler kernel called by matPlusScaler()
__global__
void matPlusScalerKernel(Matrix d_In, double scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] + scaler;
}

void matPlusScaler(Matrix In, double scaler, Matrix Out) {
	//printf("matPlusScaler()\n");
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	//printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matPlusScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);

}

// matSub kernel
__global__
void matSubKernel(Matrix d_A, Matrix d_B, Matrix d_C) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_C.elements[row*d_A.width + col] = d_A.elements[row*d_A.width + col] - d_B.elements[row*d_A.width + col];
}

void matSub(Matrix A, Matrix B, Matrix C){
	//printf("matSub()\n");
	// load A, B to device memory
	Matrix d_A;
	Matrix d_B;
	d_A.width = A.width;
	d_B.width = B.width;
	d_A.height = A.height;
	d_B.height = B.height;
	size_t size = A.width * A.height * sizeof(double);

	hipError_t err = hipMalloc(&d_A.elements, size);
	//printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	//printf("Copy A to device: %s\n", hipGetErrorString(err));

	err = hipMalloc(&d_B.elements, size);
	//printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	//printf("Copy B to device: %s\n", hipGetErrorString(err));
	
	// allocate C to device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	err = hipMalloc(&d_C.elements, size);
	//printf("CUDA malloc C: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	matSubKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read C from device memory
	err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy C off of device: %s\n", hipGetErrorString(err));

// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// matrix matTimesScaler kernel called by matTimesScaler()
__global__
void matTimesScalerKernel(Matrix d_In, double scaler, Matrix d_Out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_In.height || col >= d_In.width) return;
	int idx = row * d_In.width +  col;
	d_Out.elements[idx] = d_In.elements[idx] * scaler;
}

void matTimesScaler(Matrix In, double scaler, Matrix Out) {
	//printf("matTimesScaler()\n");
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	//printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
  d_Out.width = Out.width; d_Out.height = Out.height;
  size = Out.width * Out.height * sizeof(double);
  hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	matTimesScalerKernel<<<dimGrid, dimBlock>>>(d_In, scaler, d_Out);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

// need a better parallelized version
__global__
void maxOfMatrixRow(Matrix d_A, Matrix d_col) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	double max = d_A.elements[row*d_A.width];
	for (int col=0; col<d_A.width; col++) {
		max = (d_A.elements[row*d_A.width+col] > max)? d_A.elements[row*d_A.width+col] : max;
	}
	d_col.elements[row] = max;
}

/************************ O ************************/
// matrix ones kernel called by ones()
__global__
void onesKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > d_A.height || col > d_A.width) return;
	d_A.elements[row*d_A.width+col] = 1;
}

void ones(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	//printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	onesKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy C off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}


/************************ R ************************/

//create an m-by-n tiling of a given matrix
__global__
void repmatKernel(Matrix d_A, int m, int n, Matrix d_B) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	for(int i=0; i < m; i++) {
		for(int j=0; j < n; j++) {
			d_B.elements[(row + i*d_A.height)*d_B.width + (col + j*d_A.width)] = d_A.elements[row*d_A.width + col];
		}
	}
}

// matrix reshape kernel called by reshape()
__global__
void reshapeKernel(Matrix d_In, Matrix d_Out) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(y >= d_In.height || x >= d_In.width) return;
	int c = x * d_In.height + y;
	d_Out.elements[(c%d_Out.height)*d_Out.width+(c/d_Out.height)] = d_In.elements[(c%d_In.height)*d_In.width+(c/d_In.height)];

}

void reshape(Matrix In, Matrix Out) {
	//printf("reshape()\n");
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	//printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	reshapeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

/************************ S ************************/
__global__
void sumOfMatrixColKernel(Matrix d_A, Matrix d_row) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(col >= d_A.width) return;
	for (int row=0; row<d_A.height; row++) {
		d_row.elements[col] += d_A.elements[row*d_A.width+col];
	}
}

void sumOfMatrixCol(Matrix In, Matrix Out) {
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);
	hipError_t err = hipMalloc(&d_In.elements, size);
	//printf("CUDA malloc In: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy input matrix to device: %s\n", hipGetErrorString(err));
	
	// allocate Out in device memory
	Matrix d_Out;
	d_Out.width = Out.width; d_Out.height = Out.height;
	size = Out.width * Out.height * sizeof(double);
	hipMalloc(&d_Out.elements, size);

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE_DIM1);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x );
	sumOfMatrixColKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy output matrix off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

/************************ T ************************/

//matrix transpose kernel
__global__
void transposeKernel(Matrix d_A, Matrix d_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_B.elements[col*d_B.width+row] = d_A.elements[row*d_A.width+col];
}

void transpose(Matrix In, Matrix Out) {
	//printf("transpose()\n");
	// load In to device memory
	Matrix d_In;
	d_In.width = In.width;
	d_In.height = In.height;
	size_t size = In.width * In.height * sizeof(double);

	hipError_t err = hipMalloc(&d_In.elements, size);
	//printf("CUDA malloc In: %s\n", hipGetErrorString(err));
	hipMemcpy(d_In.elements, In.elements, size, hipMemcpyHostToDevice);
	//printf("Copy In to device: %s\n", hipGetErrorString(err));

	// allocate Out on device memory
	Matrix d_Out;
	d_Out.width = Out.width;
	d_Out.height = Out.height;
	size = d_Out.width * d_Out.height * sizeof(double);
	err = hipMalloc(&d_Out.elements, size);
	//printf("CUDA malloc d_Out: %s\n", hipGetErrorString(err));

	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (In.width + dimBlock.x - 1)/dimBlock.x, (In.height + dimBlock.y - 1)/dimBlock.y );
	transposeKernel<<<dimGrid, dimBlock>>>(d_In, d_Out);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read Out from device memory
	err = hipMemcpy(Out.elements, d_Out.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy d_Out off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_In.elements);
	hipFree(d_Out.elements);
}

/************************ Z ************************/

// matrix zeros kernel called by zeros()
__global__
void zerosKernel(Matrix d_A) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row >= d_A.height || col >= d_A.width) return;
	d_A.elements[row*d_A.width+col] = 0;
}

void zeros(Matrix A) {
	// load A to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipError_t err = hipMalloc(&d_A.elements, size);
	//printf("CUDA malloc A: %s\n", hipGetErrorString(err));	
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);	
	//printf("Copy A to device: %s\n", hipGetErrorString(err));
	
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( (A.width + dimBlock.x - 1)/dimBlock.x, (A.height + dimBlock.y - 1)/dimBlock.y );
	zerosKernel<<<dimGrid, dimBlock>>>(d_A);
	err = hipDeviceSynchronize();
	//printf("Run kernel: %s\n", hipGetErrorString(err));

	// read A from device memory
	err = hipMemcpy(A.elements, d_A.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy A off of device: %s\n",hipGetErrorString(err));

	// free device memory
	hipFree(d_A.elements);
}

